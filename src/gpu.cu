#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <string.h>
#include <cmath>
using namespace std;

/*define GPU specific variables*/
#define GPU_ID    0

#define BLOCK_Xe  512
#define BLOCK_Xv  512

// Device norm subroutine
#define blockId            (blockIdx.x)
#define threadId           (threadIdx.x)
#define isBlockMaster      (threadIdx.x==0)

/*CPU Code*/
/*I/O stuff {{{*/
FILE* SetFilePointerToData(FILE* fid,int* pcode,int* pvector_type,const char* data_name){/*{{{*/

	int found  = 0;
	const char* mddot = "md.";
	char* record_name = NULL;
	int   record_name_size;
	long long record_length;
	int record_code;       //1 to 7 number
	int vector_type   = 0; //nodal or elementary

	if(strncmp(data_name,mddot,3)!=0){
		std::cerr <<"Cannot fetch \""<<data_name<<"\" does not start with \""<<mddot<<"\"";
	}

	/*First set FILE* position to the beginning of the file: */
	fseek(fid,0,SEEK_SET);

	/*Now march through file looking for the correct data identifier: */
	for(;;){
		/*Read size of first string name: */
		if(fread(&record_name_size,sizeof(int),1,fid)==0){
			/*we have reached the end of the file. break: */
			delete record_name;
			break;
		}
		if(record_name_size<3 || record_name_size>80){
			std::cerr<<"error while looking in binary file. Found a string of size "<<record_name_size;
		}

		/*Allocate string of correct size: */
		record_name=new char[record_name_size+1];
		record_name[record_name_size]='\0';

		/*Read record_name: */
		if(fread(record_name,record_name_size*sizeof(char),1,fid)==0){
			/*we have reached the end of the file. break: */
			found=0;
			delete [] record_name;
			break;
		}
		if(strncmp(record_name,mddot,3)!=0){
			std::cerr<<"error while reading binary file: record does not start with \"md.\": "<<record_name;
		}

		/*Is this the record sought for? : */
		if(strcmp(record_name,data_name)==0){
			/*Ok, we have found the correct string. Pass the record length, and read data type code: */
			fseek(fid,sizeof(long long),SEEK_CUR);
			if(fread(&record_code,sizeof(int),1,fid)!=1) std::cerr<<"Could not read record_code";
			/*if record_code points to a vector, get its type (nodal or elementary): */
			if((5<=record_code && record_code<=7) || record_code==10){
				if(fread(&vector_type,sizeof(int),1,fid)!=1) std::cerr<<"Could not read vector_type";
			}
			found=1;
			delete [] record_name;
			break;
		}
		else{
			/*This is not the correct string, read the record length, and use it to skip this record: */
			if(fread(&record_length,sizeof(long long),1,fid)!=1) std::cerr<<"Could not read record_length";
			/*skip: */
			fseek(fid,record_length,SEEK_CUR);
			delete [] record_name;
		}
	}
	if(!found) std::cerr<<"could not find data with name \"" << data_name << "\" in binary file";

	/*Assign output pointers:*/
	*pcode=record_code;
	if(pvector_type) *pvector_type=vector_type;

	return fid;
}
/*}}}*/
void  FetchData(FILE* fid,int* pinteger,const char* data_name){/*{{{*/

	/*output: */
	int integer;
	int code;

	/*Set file pointer to beginning of the data: */
	fid=SetFilePointerToData(fid,&code,NULL,data_name);

	if(code!=2)std::cerr <<"expecting an integer for \"" << data_name<<"\"";

	/*We have to read a integer from disk. First read the dimensions of the integer, then the integer: */
	if(fread(&integer,sizeof(int),1,fid)!=1) std::cerr<<"could not read integer ";

	/*Assign output pointers: */
	*pinteger=integer;
}/*}}}*/
void  FetchData(FILE* fid,int** pmatrix,int* pM,int* pN,const char* data_name){/*{{{*/

	/*output: */
	int M,N;
	double* matrix=NULL;
	int* integer_matrix=NULL;
	int code=0;

	/*Set file pointer to beginning of the data: */
	fid=SetFilePointerToData(fid,&code,NULL,data_name);
	if(code!=5 && code!=6 && code!=7)std::cerr<<"expecting a IssmDouble, integer or boolean matrix for \""<<data_name<<"\""<<" (Code is "<<code<<")";

	/*Now fetch: */

	/*We have to read a matrix from disk. First read the dimensions of the matrix, then the whole matrix: */
	/*numberofelements: */
	if(fread(&M,sizeof(int),1,fid)!=1) std::cerr<<"could not read number of rows for matrix ";
	if(fread(&N,sizeof(int),1,fid)!=1) std::cerr<<"could not read number of columns for matrix ";

	/*Now allocate matrix: */
	if(M*N){
		matrix=new double[M*N];

		/*Read matrix on node 0, then broadcast: */
		if(fread(matrix,M*N*sizeof(double),1,fid)!=1) std::cerr<<"could not read matrix ";
	}

	/*Now cast to integer: */
	if(M*N){
		integer_matrix=new int[M*N];
		for (int i=0;i<M;i++){
			for (int j=0;j<N;j++){
				integer_matrix[i*N+j]=(int)matrix[i*N+j];
			}
		}
	}
	else{
		integer_matrix=NULL;
	}
	/*Free ressources:*/
	delete [] matrix;

	/*Assign output pointers: */
	*pmatrix=integer_matrix;
	if(pM)*pM=M;
	if(pN)*pN=N;
}/*}}}*/
void  FetchData(FILE* fid,double* pdouble,const char* data_name){/*{{{*/

	/*output: */
	double value;
	int code;

	/*Set file pointer to beginning of the data: */
	fid=SetFilePointerToData(fid,&code,NULL,data_name);

	if(code!=3)std::cerr <<"expecting a double for \"" << data_name<<"\"";

	/*We have to read a integer from disk. First read the dimensions of the integer, then the integer: */
	if(fread(&value,sizeof(double),1,fid)!=1) std::cerr<<"could not read scalar";

	/*Assign output pointers: */
	*pdouble=value;
}/*}}}*/
void  FetchData(FILE* fid,double** pmatrix,int* pM,int* pN,const char* data_name){/*{{{*/

	/*output: */
	int M,N;
	double* matrix=NULL;
	int* integer_matrix=NULL;
	int code=0;

	/*Set file pointer to beginning of the data: */
	fid=SetFilePointerToData(fid,&code,NULL,data_name);
	if(code!=5 && code!=6 && code!=7)std::cerr<<"expecting a IssmDouble, integer or boolean matrix for \""<<data_name<<"\""<<" (Code is "<<code<<")";

	/*Now fetch: */

	/*We have to read a matrix from disk. First read the dimensions of the matrix, then the whole matrix: */
	/*numberofelements: */
	if(fread(&M,sizeof(int),1,fid)!=1) std::cerr<<"could not read number of rows for matrix ";
	if(fread(&N,sizeof(int),1,fid)!=1) std::cerr<<"could not read number of columns for matrix ";

	/*Now allocate matrix: */
	if(M*N){
		matrix=new double[M*N];

		/*Read matrix on node 0, then broadcast: */
		if(fread(matrix,M*N*sizeof(double),1,fid)!=1) std::cerr<<"could not read matrix ";
	}

	/*Assign output pointers: */
	*pmatrix=matrix;
	if(pM)*pM=M;
	if(pN)*pN=N;
}/*}}}*/
void  WriteData(FILE* fid,double* matrix,int M,int N,const char* data_name){/*{{{*/

	/*First write enum: */
	int length=(strlen(data_name)+1)*sizeof(char);
	fwrite(&length,sizeof(int),1,fid);
	fwrite(data_name,length,1,fid);

	/*Now write time and step: */
	double time = 0.;
	int    step = 1;
	fwrite(&time,sizeof(double),1,fid);
	fwrite(&step,sizeof(int),1,fid);

	/*writing a IssmDouble array, type is 3:*/
	int type=3;
	fwrite(&type,sizeof(int),1,fid);
	fwrite(&M,sizeof(int),1,fid);
	fwrite(&N,sizeof(int),1,fid);
	fwrite(matrix,M*N*sizeof(double),1,fid);
}/*}}}*/
void  WriteData(FILE* fid,const char* string,const char* data_name){/*{{{*/

	/*First write enum: */
	int length=(strlen(data_name)+1)*sizeof(char);
	fwrite(&length,sizeof(int),1,fid);
	fwrite(data_name,length,1,fid);

	/*Now write time and step: */
	double time = 0.;
	int    step = 1;
	fwrite(&time,sizeof(double),1,fid);
	fwrite(&step,sizeof(int),1,fid);

	/*writing a string, type is 2: */
	int type=2;
	fwrite(&type,sizeof(int),1,fid);

	length=(strlen(string)+1)*sizeof(char);
	fwrite(&length,sizeof(int),1,fid);
	fwrite(string,length,1,fid);
}/*}}}*/
/*}}}*/
void NodalCoeffs(double** pareas,double** palpha,double** pbeta,int* index,double* x,double* y,int nbe){/*{{{*/

	/*Allocate output vectors*/
	double* areas = new double[nbe];
	double* alpha = new double[nbe*3];
	double* beta  = new double[nbe*3];

   /*Loop over all elements and calculate nodal function coefficients and element surface area*/
	for(int i = 0; i < nbe; i++) {
		int n1 = index[i*3+0]-1;
		int n2 = index[i*3+1]-1;
		int n3 = index[i*3+2]-1;

		double x1 = x[n1];
		double x2 = x[n2];
		double x3 = x[n3];
		double y1 = y[n1];
		double y2 = y[n2];
		double y3 = y[n3];

		double invdet = 1./(x1 * (y2 - y3) - x2 * (y1 - y3) + x3 * (y1 - y2));

		alpha[i*3+0] = invdet * (y2 - y3);
		alpha[i*3+1] = invdet * (y3 - y1);
		alpha[i*3+2] = invdet * (y1 - y2);

		beta[i*3+0] = invdet * (x3 - x2);
		beta[i*3+1] = invdet * (x1 - x3);
		beta[i*3+2] = invdet * (x2 - x1);

		areas[i] = 0.5*((x2-x1)*(y3-y1)-(y2-y1)*(x3-x1));
	}

	/*Assign output pointers*/
	*pareas = areas;
	*palpha = alpha;
	*pbeta  = beta;
}/*}}}*/
void Weights(double** pweights,int* index,double* areas,int nbe,int nbv){/*{{{*/

	/*Allocate output and initialize as 0*/
	double* weights = new double[nbv];
	for(int i = 0; i < nbv; i++) weights[i]=0.;

	/*Loop over elements*/
	for(int i = 0; i < nbe; i++){
		for(int j = 0; j < 3; j++){
			weights[index[i*3+j]-1] += areas[i];
		}
	}

	/*Assign output pointer*/
	*pweights = weights;
}/*}}}*/
void derive_xy_elem(double* dfdx_e,double* dfdy_e,double* f,int* index,double* alpha,double* beta,int nbe){/*{{{*/

	/*WARNING!! Assume that dfdx_e and dfdy_e have been properly allocated*/

	for(int i=0;i<nbe;i++){
		int n1 = index[i*3+0]-1;
		int n2 = index[i*3+1]-1;
		int n3 = index[i*3+2]-1;
		dfdx_e[i] = f[n1]*alpha[i*3+0] + f[n2]*alpha[i*3+1] + f[n3]*alpha[i*3+2];
		dfdy_e[i] = f[n1]*beta[ i*3+0] + f[n2]*beta[ i*3+1] + f[n3]*beta[ i*3+2];
	}
}/*}}}*/
void elem2node(double* f_v,double* f_e,int* index,double* areas,double* weights,int nbe,int nbv){/*{{{*/

	/*WARNING!! Assume that f_v has been properly allocated*/

	/*Reinitialize output*/
	for(int i=0;i<nbv;i++) f_v[i] = 0.;

	/*Add contributions from all elements connected to vertex i*/
	for(int i=0;i<nbe;i++){
		int n1 = index[i*3+0]-1;
		int n2 = index[i*3+1]-1;
		int n3 = index[i*3+2]-1;
		f_v[n1] += f_e[i]*areas[i];
		f_v[n2] += f_e[i]*areas[i];
		f_v[n3] += f_e[i]*areas[i];
	}

	/*Divide by sum of areas*/
	for(int i=0;i<nbv;i++) f_v[i] = f_v[i]/weights[i];

}/*}}}*/
void MeshSize(double* resolx,double* resoly,int* index,double* x,double* y,double* areas,double* weights,int nbe,int nbv){/*{{{*/

	/*Get element size along x and y directions*/
	double  xmin,xmax,ymin,ymax;
	double* dx_elem = new double[nbe];
	double* dy_elem = new double[nbe];
	for(int i=0;i<nbe;i++){
		int n1 = index[i*3+0]-1;
		int n2 = index[i*3+1]-1;
		int n3 = index[i*3+2]-1;
		xmin = min(min(x[n1],x[n2]),x[n3]);
		xmax = max(max(x[n1],x[n2]),x[n3]);
		ymin = min(min(y[n1],y[n2]),y[n3]);
		ymax = max(max(y[n1],y[n2]),y[n3]);
		dx_elem[i] = xmax - xmin;
		dy_elem[i] = ymax - ymin;
	}

	/*Average over each node*/
	elem2node(resolx,dx_elem,index,areas,weights,nbe,nbv);
	elem2node(resoly,dy_elem,index,areas,weights,nbe,nbv);

	/*Cleanup and return*/
	delete [] dx_elem;
	delete [] dy_elem;
}/*}}}*/

/*CUDA Code*/
void  clean_cuda(){ 
    hipError_t ce = hipGetLastError();
    if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset(); }
}

__global__ void PT1(double* dvxdx, double* dvydy, double* dvxdy, double* dvydx, double* vx, double* vy, double* alpha, double* beta, int* index, double* kvx, double* kvy, double* etan,  double* Helem, double* areas, bool* isice, double* Eta_nbe, int nbe){
 
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
     
    if (ix<nbe){
      /*Calculate velocity derivatives*/
        dvxdx[ix] = vx[index[ix*3+0]-1]*alpha[ix*3+0] + vx[index[ix*3+1]-1]*alpha[ix*3+1] + vx[index[ix*3+2]-1]*alpha[ix*3+2];
        dvxdy[ix] = vx[index[ix*3+0]-1]*beta [ix*3+0] + vx[index[ix*3+1]-1]*beta [ix*3+1] + vx[index[ix*3+2]-1]*beta [ix*3+2];
        dvydx[ix] = vy[index[ix*3+0]-1]*alpha[ix*3+0] + vy[index[ix*3+1]-1]*alpha[ix*3+1] + vy[index[ix*3+2]-1]*alpha[ix*3+2];
        dvydy[ix] = vy[index[ix*3+0]-1]*beta [ix*3+0] + vy[index[ix*3+1]-1]*beta [ix*3+1] + vy[index[ix*3+2]-1]*beta [ix*3+2];
    
    Eta_nbe[ix] = etan[ix]*areas[ix];

    
    /*Skip if no ice*/
        if (isice[ix]){
            /*Viscous Deformation*/
            double eta_e = etan[ix];
            double eps_xx = dvxdx[ix];
            double eps_yy = dvydy[ix];
            double eps_xy = .5 * (dvxdy[ix] + dvydx[ix]);
            for (int i = 0; i < 3; i++){
                kvx[ix * 3 + i] = 2 * Helem[ix] * eta_e * (2 * eps_xx + eps_yy) * alpha[ix * 3 + i] * areas[ix] + 2 * Helem[ix] * eta_e * eps_xy * beta[ix * 3 + i] * areas[ix];
                kvy[ix * 3 + i] = 2 * Helem[ix] * eta_e * eps_xy * alpha[ix * 3 + i] * areas[ix] +  2 * Helem[ix] * eta_e * (2 * eps_yy + eps_xx) * beta[ix * 3 + i] * areas[ix];
            }
        }//isice loop
    }  //ix<nbe loop
}  

//Moving to the next kernel, as kvx cannot be defined and updated in the same kernel
__global__ void PT2_x(double* kvx, double* groundedratio, double* areas, int* index, double* alpha2, double* vx, bool* isice,  int nbe){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    /*Add basal friction*/
    if (ix<nbe){
        if (isice[ix]){
            if (groundedratio[ix] > 0.){
                int n3 = ix * 3;
                double gr_a = groundedratio[ix] * areas[ix];
                for (int k = 0; k < 3; k++){
                    for (int i = 0; i < 3; i++){
                        int i_index = index[n3 + i] - 1;
                        double gr_a_alpha2 = gr_a * alpha2[i_index];
                        for (int j = 0; j < 3; j++){
                           int j_index = index[n3 + j] - 1;
                           double gr_a_alpha2_vx = gr_a_alpha2 * vx[j_index];
                   
                           // printf("%d, %f, %f, %d, %f \n", ix, gr_a, gr_a_alpha2, j_index, gr_a_alpha2_vx);
                           if (i == j && j == k){
                                kvx[n3 + k] =  kvx[n3 + k] + gr_a_alpha2_vx / 10.;
              
                            } else if ((i!=j) && (j!=k) && (k!=i)){
                                kvx[n3 + k] =  kvx[n3 + k] + gr_a_alpha2_vx / 60.;

                            } else{
                                kvx[n3 + k] =  kvx[n3 + k] + gr_a_alpha2_vx / 30.;
 
                           }
                        }
                    }
                }
            }//groundedratio loop
        }//isice loop
    }//nbe loop 
}

__global__ void PT2_y(double* kvy, double* groundedratio, double* areas, int* index, double* alpha2, double* vy, bool* isice,  int nbe){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    /*Add basal friction*/
    if (ix<nbe){
        if (isice[ix]){
            if (groundedratio[ix] > 0.){
                int n3 = ix * 3;
                double gr_a = groundedratio[ix] * areas[ix];
                for (int k = 0; k < 3; k++){
                    for (int i = 0; i < 3; i++){
                        int i_index = index[n3 + i] - 1;
                        double gr_a_alpha2 = gr_a * alpha2[i_index];
                        for (int j = 0; j < 3; j++){
                           int j_index = index[n3 + j] - 1;
        
                           double gr_a_alpha2_vy = gr_a_alpha2 * vy[j_index];
                           // printf("%d, %f, %f, %d, %f \n", ix, gr_a, gr_a_alpha2, j_index, gr_a_alpha2_vx);
                           if (i == j && j == k){
                   
                                kvy[n3 + k] =  kvy[n3 + k] + gr_a_alpha2_vy / 10.;
                            } else if ((i!=j) && (j!=k) && (k!=i)){
                
                                kvy[n3 + k] =  kvy[n3 + k] + gr_a_alpha2_vy / 60.;
                            } else{
                
                                kvy[n3 + k] =  kvy[n3 + k] + gr_a_alpha2_vy / 30.;
                           }
                        }
                    }
                }
            }//groundedratio loop
        }//isice loop
    }//nbe loop 
}
//Moving to the next kernel::cannot update kvx and perform indirect access, lines 474 and 475, in the same kernel//
__global__ void PT3(double* kvx, double* kvy, double* Eta_nbe, double* areas, double* eta_nbv, int* index, int* connectivity, int* columns, double* weights, double* ML, double* KVx, double* KVy, double* Fvx, double* Fvy, double* dVxdt, double* dVydt, double* resolx, double* resoly, double* H, double* vx, double* vy, double* spcvx, double* spcvy, double rho, double damp, double relaxation, double eta_b, int nbv){ 

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
 
    __shared__ volatile double ResVx;
    //double ResVx;
    double ResVy;
    double dtVx;
    double dtVy;
	
    if (ix<nbv){
        KVx[ix] = 0.;
        KVy[ix] = 0.;
    
    
        for(int j=0;j<8;j++){
            if (connectivity[(ix * 8 + j)] != 0){
                KVx[ix] = KVx[ix] + kvx[((connectivity[(ix * 8 + j)])-1) *3 + ((columns[(ix * 8 + j)]))] ;
                KVy[ix] = KVy[ix] + kvy[((connectivity[(ix * 8 + j)])-1) *3 + ((columns[(ix * 8 + j)]))] ;
            }
        }
    
 
    
        for (int j = 0; j < 8; j++){
            if (connectivity[(ix * 8 + j)] != 0){      
                eta_nbv[ix] = eta_nbv[ix] + Eta_nbe[connectivity[(ix * 8 + j)]-1];
            }
        }
 
  eta_nbv[ix] =eta_nbv[ix]/weights[ix];
    
   

        /*1. Get time derivative based on residual (dV/dt)*/
        ResVx =  1./(rho*max(80.0,H[ix])*ML[ix])*(-KVx[ix] + Fvx[ix]); //rate of velocity in the x, equation 23
        ResVy =  1./(rho*max(80.0,H[ix])*ML[ix])*(-KVy[ix] + Fvy[ix]); //rate of velocity in the y, equation 24
        
        // dVxdt[ix] = dVxdt[ix]*(1.-damp/20.) + ResVx;
        // dVydt[ix] = dVydt[ix]*(1.-damp/20.) + ResVy;
        dVxdt[ix] = dVxdt[ix]*damp + ResVx;
        dVydt[ix] = dVydt[ix]*damp + ResVy;

        /*2. Explicit CFL time step for viscous flow, x and y directions*/
        dtVx = rho*resolx[ix]*resol[ix]/(4*eta_nbv[ix]*(1.+eta_b)*4.1);
        dtVy = rho*resoly[ix]*resol[ix]/(4*eta_nbv[ix]*(1.+eta_b)*4.1);
        // dtVx = rho*pow(resolx[ix],2)/(4*H[ix]*eta_nbv[ix]*(1.+eta_b)*4.1)*relaxation;
        // dtVy = rho*pow(resoly[ix],2)/(4*H[ix]*eta_nbv[ix]*(1.+eta_b)*4.1)*relaxation;     

        /*3. velocity update, vx(new) = vx(old) + change in vx, Similarly for vy*/
        vx[ix] = vx[ix] + relaxation*dVxdt[ix]*dtVx;
        vy[ix] = vy[ix] + relaxation*dVydt[ix]*dtVy;

        /*Apply Dirichlet boundary condition*/
        if (!isnan(spcvx[ix])){
            vx[ix]    = spcvx[ix];
            dVxdt[ix] = 0.;
        }
        if (!isnan(spcvy[ix])){
            vy[ix]    = spcvy[ix];
            dVydt[ix] = 0.;
        }
    }
    
}

__global__ void PT4(double* etan, double* dvxdx, double* dvydy, double* dvxdy, double* dvydx, double* rheology_B, double n_glen, bool* isice, double eta_0, double rele, int nbe){
 
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < nbe){
        double  eps_xx = dvxdx[ix];
        double  eps_yy = dvydy[ix];
        double  eps_xy = .5*(dvxdy[ix]+dvydx[ix]);
        double  EII2 = eps_xx*eps_xx + eps_yy*eps_yy + eps_xy*eps_xy + eps_xx*eps_yy;
        double  eta_it = 1.e+14/2.0;

        if (EII2>0.) eta_it = rheology_B[ix]/(2*pow(EII2,(n_glen-1.)/(2*n_glen)));
    
        if (isice[ix]) etan[ix]  = min(exp(rele*log(eta_it) + (1-rele)*log(etan[ix])),eta_0*1e5);
    }
}

// Find the norm of an array
__shared__ volatile double block_normval;
__global__ void __device_norm_d(double* A, int nbv, double* device_normval){
   
    double thread_normval=0.0;
   
    int ix  = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    // find the normval for each block
    if (ix<nbv){ thread_normval = A[ix]*A[ix]; }
    if (threadIdx.x==0){ block_normval=0.0; }
    __syncthreads();
    for (int i=0; i < (BLOCK_Xv); i++){
        if (i==threadIdx.x){ block_normval = block_normval + thread_normval; }
        __syncthreads();
    }
    device_normval[blockIdx.x] = block_normval;
}

#define __device_normx(dVxdt)   __device_norm_d<<<gridv, blockv>>>(d_dVxdt, nbv, d_device_normvalx); \
                                hipMemcpy(device_normvalx, d_device_normvalx, GRID_Xv*sizeof(double), hipMemcpyDeviceToHost); \
                                double device_NORMx = 0.0;                                     \
                                for (int i=0; i < (GRID_Xv); i++){                            \
                                    device_NORMx = device_NORMx + device_normvalx[i];          \
                                }                                                              \
                                device_NORMx = (double)1.0/((double)nbv)*sqrt(device_NORMx);

#define __device_normy(dVydt)   __device_norm_d<<<gridv, blockv>>>(d_dVydt, nbv, d_device_normvaly); \
                                hipMemcpy(device_normvaly, d_device_normvaly, GRID_Xv*sizeof(double), hipMemcpyDeviceToHost); \
                                double device_NORMy = 0.0;                                     \
                                for (int i=0; i < (GRID_Xv); i++){                            \
                                    device_NORMy = device_NORMy + device_normvaly[i];          \
                                }                                                              \
                                device_NORMy = (double)1.0/((double)nbv)*sqrt(device_NORMy);

// Find the max of an array
__shared__ volatile double block_maxval;
__global__ void __device_max_d(double* A, int nbv, double* device_maxval){
   
    double thread_maxval=0.0;
   
    int ix  = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    // find the maxval for each block
    if (ix<nbv){ thread_maxval = abs(A[ix]); }
    if (threadIdx.x==0){ block_maxval=0.0; }
    __syncthreads();
    for (int i=0; i < (BLOCK_Xv); i++){
        if (i==threadIdx.x){ block_maxval = max(block_maxval, thread_maxval); }
        __syncthreads();
    }
    device_maxval[blockIdx.x] = block_maxval;
}

#define __device_max_x(dVxdt)   __device_max_d<<<gridv, blockv>>>(d_dVxdt, nbv, d_device_maxvalx); \
                                hipMemcpy(device_maxvalx, d_device_maxvalx, GRID_Xv*sizeof(double), hipMemcpyDeviceToHost); \
                                double device_MAXx = 0.0;                                     \
                                for (int i=0; i < (GRID_Xv); i++){                            \
                                    device_MAXx = max(device_MAXx, device_maxvalx[i]);        \
                                }

#define __device_max_y(dVydt)   __device_max_d<<<gridv, blockv>>>(d_dVydt, nbv, d_device_maxvaly); \
                                hipMemcpy(device_maxvaly, d_device_maxvaly, GRID_Xv*sizeof(double), hipMemcpyDeviceToHost); \
                                double device_MAXy = 0.0;                                     \
                                for (int i=0; i < (GRID_Xv); i++){                            \
                                    device_MAXy = max(device_MAXy, device_maxvaly[i]);        \
  
 // timer
#include "sys/time.h"
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }

/*Main*/
int main(){/*{{{*/

	/*Open input binary file*/
	const char* inputfile  = "./JKS.bin";
	const char* outputfile = "./output.outbin";
	FILE* fid = fopen(inputfile,"rb");
	if(fid==NULL) std::cerr<<"could not open file " << inputfile << " for binary reading or writing";

	/*Get All we need from binary file*/
	int    nbe,nbv,M,N;
	double g,rho,rho_w,yts;
	int    *index           = NULL;
	double *spcvx           = NULL;
	double *spcvy           = NULL;
	double *x               = NULL;
	double *y               = NULL;
	double *H               = NULL;
	double *surface         = NULL;
	double *base            = NULL;
	double *ice_levelset    = NULL;
	double *ocean_levelset  = NULL;
	double *rheology_B_temp = NULL;
	double *vx              = NULL;
	double *vy              = NULL;
	double *friction        = NULL;
	FetchData(fid,&nbe,"md.mesh.numberofelements");
	FetchData(fid,&nbv,"md.mesh.numberofvertices");
	FetchData(fid,&g,"md.constants.g");
	FetchData(fid,&rho,"md.materials.rho_ice");
	FetchData(fid,&rho_w,"md.materials.rho_water");
	FetchData(fid,&yts,"md.constants.yts");
	FetchData(fid,&index,&M,&N,"md.mesh.elements");
	FetchData(fid,&spcvx,&M,&N,"md.stressbalance.spcvx");
	FetchData(fid,&spcvy,&M,&N,"md.stressbalance.spcvy");
	FetchData(fid,&x,&M,&N,"md.mesh.x");
	FetchData(fid,&y,&M,&N,"md.mesh.y");
	FetchData(fid,&H,&M,&N,"md.geometry.thickness");
	FetchData(fid,&surface,&M,&N,"md.geometry.surface");
	FetchData(fid,&base,&M,&N,"md.geometry.base");
	FetchData(fid,&ice_levelset,&M,&N,"md.mask.ice_levelset");
	FetchData(fid,&ocean_levelset,&M,&N,"md.mask.ocean_levelset");
	FetchData(fid,&rheology_B_temp,&M,&N,"md.materials.rheology_B");
	FetchData(fid,&vx,&M,&N,"md.initialization.vx");
	FetchData(fid,&vy,&M,&N,"md.initialization.vy");
	FetchData(fid,&friction,&M,&N,"md.friction.coefficient");

	/*Close input file*/
	if(fclose(fid)!=0) std::cerr<<"could not close file " << inputfile;

	/*Constants*/
	double n_glen    = 3.;
	double damp      = 0.96;  //may need to change this depending on the glacier model and the spatial resolution
	double rele      = 1e-1;
	double eta_b     = 0.5;
	double eta_0     = 1.e+14/2.;
	int    niter     = 5e6;
	int    nout_iter = 1000;
        double epsi       = 3.171e-7;
        double relaxation = 0.7;
    
     // Ceiling division to get the close to optimal GRID size
    unsigned int GRID_Xe = 1 + ((nbe - 1) / BLOCK_Xe);
    unsigned int GRID_Xv = 1 + ((nbv - 1) / BLOCK_Xv);

    std::cout<<"GRID_Xe="<<GRID_Xe<<std::endl;
    std::cout<<"GRID_Xv="<<GRID_Xv<<std::endl;

    // Set up GPU
    int gpu_id=-1;
    dim3 gridv, blockv;
    dim3 gride, blocke;
    blockv.x = BLOCK_Xv; gridv.x = GRID_Xv;
    blocke.x = BLOCK_Xe; gride.x = GRID_Xe;
    gpu_id = GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n", gpu_id);
    //hipSetDevice  selects the device, set the gpu id you selected
	
	
	/*Initial guesses (except vx and vy that we already loaded)*/
	double* etan = new double[nbe];
	for(int i=0;i<nbe;i++) etan[i] = 1.e+14;
	double* dVxdt = new double[nbv];
	for(int i=0;i<nbv;i++) dVxdt[i] = 0.;
	double* dVydt = new double[nbv];
	for(int i=0;i<nbv;i++) dVydt[i] = 0.;

	/*Manage derivatives once for all*/
	double* alpha   = NULL;
	double* beta    = NULL;
	double* areas   = NULL;
	double* weights = NULL;
	NodalCoeffs(&areas,&alpha,&beta,index,x,y,nbe);
	Weights(&weights,index,areas,nbe,nbv);

	/*MeshSize*/
	double* resolx = new double[nbv];
	double* resoly = new double[nbv];
	MeshSize(resolx,resoly,index,x,y,areas,weights,nbe,nbv);

	/*Physical properties once for all*/
	double* dsdx = new double[nbe];
	double* dsdy = new double[nbe];
	derive_xy_elem(dsdx,dsdy,surface,index,alpha,beta,nbe);
	double* Helem      = new double[nbe];
	double* rheology_B = new double[nbe];
	for(int i=0;i<nbe;i++){
		Helem[i]      = 1./3. * (H[index[i*3+0]-1] + H[index[i*3+1]-1] + H[index[i*3+2]-1]);
		rheology_B[i] = 1./3. * (rheology_B_temp[index[i*3+0]-1] + rheology_B_temp[index[i*3+1]-1] + rheology_B_temp[index[i*3+2]-1]);
	}
    //Initial viscosity//
    double* dvxdx   = new double[nbe];
    double* dvxdy   = new double[nbe];
    double* dvydx   = new double[nbe];
    double* dvydy   = new double[nbe];

  
    derive_xy_elem(dvxdx,dvxdy,vx,index,alpha,beta,nbe);
    derive_xy_elem(dvydx,dvydy,vy,index,alpha,beta,nbe);

    for(int i=0;i<nbe;i++){
        double eps_xx = dvxdx[i];
        double eps_yy = dvydy[i];
        double eps_xy = .5*(dvxdy[i]+dvydx[i]);
        double EII2 = pow(eps_xx,2) + pow(eps_yy,2) + pow(eps_xy,2) + eps_xx*eps_yy;
        double eta_it = 1.e+14/2.;
        if(EII2>0.) eta_it = rheology_B[i]/(2*pow(EII2,(n_glen-1.)/(2*n_glen)));

        etan[i] = min(eta_it,eta_0*1e5);
        if(isnan(etan[i])){ std::cerr<<"Found NaN in etan[i]"; return 1;}
    }
	
	
	    /*Linear integration points order 3*/
    double wgt3[] = { 0.555555555555556, 0.888888888888889, 0.555555555555556 };
    double xg3[]  = {-0.774596669241483, 0.000000000000000, 0.774596669241483 };

	
	/*Compute RHS amd ML once for all*/
	double* ML            = new double[nbv];
	double* Fvx           = new double[nbv];
	double* Fvy           = new double[nbv];
	double* groundedratio = new double[nbe];
	bool*   isice         = new bool[nbe];     
        double level[3];    
	for(int i=0;i<nbv;i++){
		ML[i]  = 0.;
		Fvx[i] = 0.;
		Fvy[i] = 0.;
	}
	for(int n=0;n<nbe;n++){
		/*Lumped mass matrix*/
		for(int i=0;i<3;i++){
			for(int j=0;j<3;j++){
				// \int_E phi_i * phi_i dE = A/6 and % \int_E phi_i * phi_j dE = A/12
				if(i==j)
				 ML[index[n*3+j]-1] += areas[n]/6.;
				else
				 ML[index[n*3+j]-1] += areas[n]/12.;
			}
		}
		
		/*Is there ice at all in the current element?*/
        level[0] = ice_levelset[index[n*3+0]-1];
        level[1] = ice_levelset[index[n*3+1]-1];
        level[2] = ice_levelset[index[n*3+2]-1];
        if (level[0]<0 || level[1]<0 || level[2]<0){
            isice[n] = true;
        }
        else{
            isice[n] = false;
            for(int i=0;i<3;i++){
                vx[index[n*3+i]-1] = 0.;
                vy[index[n*3+i]-1] = 0.;
            }
            continue;
        }
		
		/*RHS, 'F ' in equation 22 (Driving Stress)*/
		for(int i=0;i<3;i++){
			for(int j=0;j<3;j++){
				if(i==j){
					Fvx[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdx[n]*areas[n]/6.;
					Fvy[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdy[n]*areas[n]/6.;
				}
				else{
					Fvx[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdx[n]*areas[n]/12.;
					Fvy[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdy[n]*areas[n]/12.;
				}
			}
		}
	}

	/*RHS (Water pressure at the ice front)*/

	for(int n=0;n<nbe;n++){
		/*Determine if there is an ice front there*/
		level[0] = ice_levelset[index[n*3+0]-1];
		level[1] = ice_levelset[index[n*3+1]-1];
		level[2] = ice_levelset[index[n*3+2]-1];
		int count = 0;
		for(int i=0;i<3;i++) if(level[i]<0.) count++;
		if(count==1){

			/*Ok this element has an ice front, get indices of the 2 vertices*/
			int seg1[2] = {index[n*3+0]-1,index[n*3+1]-1};
			int seg2[2] = {index[n*3+1]-1,index[n*3+2]-1};
			int seg3[2] = {index[n*3+2]-1,index[n*3+0]-1};
			int pairids[2];
			if(ice_levelset[seg1[0]]>=0 && ice_levelset[seg1[1]]>=0){
				pairids[0] = seg1[0]; pairids[1] = seg1[1];
			}
			else if (ice_levelset[seg2[0]]>=0 && ice_levelset[seg2[1]]>=0){
				pairids[0] = seg2[0]; pairids[1] = seg2[1];
			}
			else if (ice_levelset[seg3[0]]>=0 && ice_levelset[seg3[1]]>=0){
				pairids[0] = seg3[0]; pairids[1] = seg3[1];
			}
			else{
				std::cerr<<"case not supported";
			}

			/*Get normal*/
			double len = sqrt(pow(x[pairids[1]]-x[pairids[0]],2) + pow(y[pairids[1]]-y[pairids[0]],2) );
			double nx  = +(y[pairids[1]]-y[pairids[0]])/len;
			double ny  = -(x[pairids[1]]-x[pairids[0]])/len;

			 /*RHS*/
            for(int gg=0;gg<2;gg++){
                double phi1 = (1.0 -xg3[gg])/2.;
                double phi2 = (1.0 +xg3[gg])/2.;
                double bg = base[pairids[0]]*phi1 + base[pairids[1]]*phi2;
                double Hg = H[pairids[0]]*phi1 + H[pairids[1]]*phi2;
                bg = min(bg,0.0);
                Fvx[pairids[0]] = Fvx[pairids[0]] +wgt3[gg]/2*1/2*(-rho_w*g* pow(bg,2)+rho*g*pow(Hg,2))*nx*len*phi1;
                Fvx[pairids[1]] = Fvx[pairids[1]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*nx*len*phi2;
                Fvy[pairids[0]] = Fvy[pairids[0]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*ny*len*phi1;
                Fvy[pairids[1]] = Fvy[pairids[1]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*ny*len*phi2;
            } 
        }

		/*One more thing in this element loop: prepare groundedarea needed later for the calculation of basal friction*/
		level[0] = ocean_levelset[index[n*3+0]-1];
		level[1] = ocean_levelset[index[n*3+1]-1];
		level[2] = ocean_levelset[index[n*3+2]-1];
		if(level[0]>=0. && level[1]>=0. && level[2]>=0.){
			/*Completely grounded*/
			groundedratio[n]=1.;
		}
		else if(level[0]<=0. && level[1]<=0. && level[2]<=0.){
			/*Completely floating*/
			groundedratio[n]=0.;
		}
		else{
			/*Partially floating,*/
			double s1,s2;
			if(level[0]*level[1]>0){/*Nodes 0 and 1 are similar, so points must be found on segment 0-2 and 1-2*/
				s1=level[2]/(level[2]-level[1]);
				s2=level[2]/(level[2]-level[0]);
			}
			else if(level[1]*level[2]>0){ /*Nodes 1 and 2 are similar, so points must be found on segment 0-1 and 0-2*/
				s1=level[0]/(level[0]-level[1]);
				s2=level[0]/(level[0]-level[2]);
			}
			else if(level[0]*level[2]>0){/*Nodes 0 and 2 are similar, so points must be found on segment 1-0 and 1-2*/
				s1=level[1]/(level[1]-level[0]);
				s2=level[1]/(level[1]-level[2]);
			}
			else{
				std::cerr<<"should not be here...";
			}

			if(level[0]*level[1]*level[2]>0.){
				/*two nodes floating, inner triangle is grounded*/
				groundedratio[n]= s1*s2;
			}
			else{
				/*one node floating, inner triangle is floating*/
				groundedratio[n]= (1.-s1*s2);
			}
		}
	}

	/*Finally add calculation of friction coefficient*/
	double* alpha2 = new double[nbv];
	for(int i=0;i<nbv;i++){
		/*Compute effective pressure*/
		double p_ice   = g*rho*H[i];
		double p_water = -rho_w*g*base[i];
		double Neff    = p_ice - p_water;
		if(Neff<0.) Neff=0.;

		/*Compute alpha2*/
		alpha2[i] = pow(friction[i],2)*Neff;
	}

	
   //prepare head and next vectors for chain algorithm, at this point we have not seen any of the elements, so just set the head to -1 (=stop)
    int* head = new int[nbv];
    int* next  = new int[3*nbe];
    for(int i=0;i<nbv;i++) head[i] = -1;

    //Now construct the chain
    for(int k=0;k<nbe;k++){
        for(int j=0;j<3;j++){
            int i;
            int p = 3*k+j;       //unique linear index of current vertex in index
            i = index[p];
            next[p] = head[i - 1];
            head[i -1] = p + 1;
         //   std::cout << "i = " << index[p] << "head = " << head[i] <<"next = " << next[p] << std::endl;
        }
    }
  //  for(int i=0;i<nbe*3;i++) { std::cout << "next = " << next[i] << std::endl;}
    //Note: Index array starts at 0, but the node# starts at 1
    //Now we can construct the connectivity matrix
    int MAXCONNECT = 8;
    int* connectivity = new int[nbv*MAXCONNECT];
    int* columns = new int[nbv*MAXCONNECT];

    for(int i=0;i<nbv;i++) {

        /*Go over all of the elements connected to node I*/
        int count = 0;
        int p=head[i];

        //for (int p = head[i]; p != -1; p = next[p]) {
          while (p!= -1) {

              int k = p / 3 + 1;     //”row" in index
              int j = (p % 3) - 1;   //"column" in index

              if (j==-1) {
                  j=2;
              k= k -1;}

             //  std::cout << "p = " << p<< "k = " << k << ", j = " << j <<", i =" <<i + 1 <<", index =" <<index[p-1] << std::endl;

               //sanity check
            if (index[p-1] !=i+1) {
                std::cout << "Error occurred"  << std::endl;;
            }

            //enter element in connectivity matrix
            connectivity[i * MAXCONNECT + count] = k;
            columns[i * MAXCONNECT + count] = j;
            count++;
            p = next[p-1];
        }
    }
	
    double* device_normvalx = new double[GRID_Xv];
    double* device_normvaly = new double[GRID_Xv];
    for(int i=0;i<GRID_Xv;i++) device_normvalx[i] = 0.;
    for(int i=0;i<GRID_Xv;i++) device_normvaly[i] = 0.;

    double* device_maxvalx = new double[GRID_Xv];
    double* device_maxvaly = new double[GRID_Xv];
    for(int i=0;i<GRID_Xv;i++) device_maxvalx[i] = 0.;
    for(int i=0;i<GRID_Xv;i++) device_maxvaly[i] = 0.;
	
	
   /*------------ now copy all relevant vectors from host to device ---------------*/

	int *d_index = NULL;
	hipMalloc(&d_index, nbe*3*sizeof(int));
	hipMemcpy(d_index, index, nbe*3*sizeof(int), hipMemcpyHostToDevice);

	double *d_vx;
	hipMalloc(&d_vx, nbv*sizeof(double));
	hipMemcpy(d_vx, vx, nbv*sizeof(double), hipMemcpyHostToDevice);  

	double *d_vy;
	hipMalloc(&d_vy, nbv*sizeof(double));
	hipMemcpy(d_vy, vy, nbv*sizeof(double), hipMemcpyHostToDevice);  

	double *d_alpha;
	hipMalloc(&d_alpha, nbe*3*sizeof(double));
	hipMemcpy(d_alpha, alpha, nbe*3*sizeof(double), hipMemcpyHostToDevice);

	double *d_beta;
	hipMalloc(&d_beta, nbe*3*sizeof(double));
	hipMemcpy(d_beta, beta, nbe*3*sizeof(double), hipMemcpyHostToDevice);

	double *d_etan;
	hipMalloc(&d_etan, nbe*sizeof(double));
	hipMemcpy(d_etan, etan, nbe*sizeof(double), hipMemcpyHostToDevice);  

	double *d_rheology_B;
	hipMalloc(&d_rheology_B, nbe*sizeof(double));
	hipMemcpy(d_rheology_B, rheology_B, nbe*sizeof(double), hipMemcpyHostToDevice); 

	double *d_Helem;
	hipMalloc(&d_Helem, nbe*sizeof(double));
	hipMemcpy(d_Helem, Helem, nbe*sizeof(double), hipMemcpyHostToDevice); 

	double *d_areas;
	hipMalloc(&d_areas, nbe*sizeof(double));
	hipMemcpy(d_areas, areas, nbe*sizeof(double), hipMemcpyHostToDevice); 

	double *d_weights;
	hipMalloc(&d_weights, nbv*sizeof(double));
	hipMemcpy(d_weights, weights, nbv*sizeof(double), hipMemcpyHostToDevice);  

	double *d_ML;
	hipMalloc(&d_ML, nbv*sizeof(double));
	hipMemcpy(d_ML, ML, nbv*sizeof(double), hipMemcpyHostToDevice);  

	double *d_Fvx;
	hipMalloc(&d_Fvx, nbv*sizeof(double));
	hipMemcpy(d_Fvx, Fvx, nbv*sizeof(double), hipMemcpyHostToDevice); 

	double *d_Fvy;
	hipMalloc(&d_Fvy, nbv*sizeof(double));
	hipMemcpy(d_Fvy, Fvy, nbv*sizeof(double), hipMemcpyHostToDevice); 

	double *d_dVxdt;
	hipMalloc(&d_dVxdt, nbv*sizeof(double));
	hipMemcpy(d_dVxdt, dVxdt, nbv*sizeof(double), hipMemcpyHostToDevice); 

	double *d_dVydt;
	hipMalloc(&d_dVydt, nbv*sizeof(double));
	hipMemcpy(d_dVydt, dVydt, nbv*sizeof(double), hipMemcpyHostToDevice); 

	double *d_resolx;
	hipMalloc(&d_resolx, nbv*sizeof(double));
	hipMemcpy(d_resolx, resolx, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_resoly;
	hipMalloc(&d_resoly, nbv*sizeof(double));
	hipMemcpy(d_resoly, resoly, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_H;
	hipMalloc(&d_H, nbv*sizeof(double));
	hipMemcpy(d_H, H, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_spcvx;
	hipMalloc(&d_spcvx, nbv*sizeof(double));
	hipMemcpy(d_spcvx, spcvx, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_spcvy;
	hipMalloc(&d_spcvy, nbv*sizeof(double));
	hipMemcpy(d_spcvy, spcvy, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_alpha2;
	hipMalloc(&d_alpha2, nbv*sizeof(double));
	hipMemcpy(d_alpha2, alpha2, nbv*sizeof(double), hipMemcpyHostToDevice);

	double *d_groundedratio;
	hipMalloc(&d_groundedratio, nbe*sizeof(double));
	hipMemcpy(d_groundedratio, groundedratio, nbe*sizeof(double), hipMemcpyHostToDevice);
	
        bool *d_isice;
        hipMalloc(&d_isice, nbe*sizeof(bool));
        hipMemcpy(d_isice, isice, nbe*sizeof(bool), hipMemcpyHostToDevice);
    
    int *d_connectivity = NULL;
    hipMalloc(&d_connectivity, nbv*8*sizeof(int));
    hipMemcpy(d_connectivity, connectivity, nbv*8*sizeof(int), hipMemcpyHostToDevice);

    int *d_columns = NULL;
    hipMalloc(&d_columns, nbv*8*sizeof(int));
    hipMemcpy(d_columns, columns, nbv*8*sizeof(int), hipMemcpyHostToDevice);
        
	
        double* d_device_normvalx = NULL;
        hipMalloc(&d_device_normvalx, GRID_Xv*sizeof(double));
        hipMemcpy(d_device_normvalx, device_normvalx, GRID_Xv*sizeof(double), hipMemcpyHostToDevice);


        double* d_device_normvaly = NULL;
        hipMalloc(&d_device_normvaly, GRID_Xv*sizeof(double));
        hipMemcpy(d_device_normvaly, device_normvaly, GRID_Xv*sizeof(double), hipMemcpyHostToDevice);
	
	
    double* d_device_maxvalx = NULL;
    hipMalloc(&d_device_maxvalx, GRID_Xv*sizeof(double));
    hipMemcpy(d_device_maxvalx, device_maxvalx, GRID_Xv*sizeof(double), hipMemcpyHostToDevice);

    double* d_device_maxvaly = NULL;
    hipMalloc(&d_device_maxvaly, GRID_Xv*sizeof(double));
    hipMemcpy(d_device_maxvaly, device_maxvaly, GRID_Xv*sizeof(double), hipMemcpyHostToDevice); 
   /*------------ allocate relevant vectors on host (GPU)---------------*/

	//double *dvxdx = NULL;
	hipMalloc(&dvxdx,nbe*sizeof(double));

	//double *dvxdy = NULL;
	hipMalloc(&dvxdy, nbe*sizeof(double));

	//double *dvydx = NULL;
	hipMalloc(&dvydx, nbe*sizeof(double));

	//double *dvydy = NULL;
	hipMalloc(&dvydy, nbe*sizeof(double));

	double *KVx = NULL;
	hipMalloc(&KVx, nbv*sizeof(double));

	double *KVy = NULL;
	hipMalloc(&KVy, nbv*sizeof(double));

	double *eta_nbv = NULL;
        hipMalloc(&eta_nbv, nbv*sizeof(double));

        double *Eta_nbe = NULL;
        hipMalloc(&Eta_nbe, nbe*3*sizeof(double));       

        double *kvx = NULL;
	hipMalloc(&kvx, nbe*3*sizeof(double));

	double *kvy = NULL;
	hipMalloc(&kvy, nbe*3*sizeof(double));
	
	
	  //Creating CUDA streams
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
	
	   // Perf
    double time_s = 0.0;
    double mem = (double)1e-9*(double)nbe*sizeof(double);
    int nIO = 10;

	/*Main loop*/
	int iter;
	double iterror;
	for(iter=1;iter<=niter;iter++){

		      if (iter==11) tic();

        PT1<<<gride, blocke>>>(dvxdx, dvydy, dvxdy, dvydx, d_vx, d_vy, d_alpha, d_beta, d_index, kvx, kvy, d_etan, d_Helem, d_areas, d_isice, Eta_nbe, nbe);
        hipDeviceSynchronize();     

        PT2_x<<<gride, blocke, 0, stream1>>>(kvx, d_groundedratio, d_areas, d_index, d_alpha2, d_vx,  d_isice, nbe);
        hipDeviceSynchronize();
        
        PT2_y<<<gride, blocke, 0, stream2>>>(kvy, d_groundedratio, d_areas, d_index, d_alpha2, d_vy, d_isice, nbe);
        hipDeviceSynchronize();
        	
        PT3<<<gridv, blockv>>>(kvx, kvy, Eta_nbe, d_areas, eta_nbv, d_index, d_connectivity, d_columns, d_weights, d_ML, KVx, KVy, d_Fvx, d_Fvy, d_dVxdt, d_dVydt, d_resolx, d_resoly, d_H, d_vx, d_vy, d_spcvx, d_spcvy, rho, damp, relaxation, eta_b, nbv);   
        hipDeviceSynchronize();
        
        PT4<<<gride, blocke>>>(d_etan, dvxdx, dvydy, dvxdy, dvydx, d_rheology_B, n_glen, d_isice, eta_0, rele, nbe);
        hipDeviceSynchronize();

        if ((iter % nout_iter) == 0){
            /*Get final error estimate*/
            __device_max_x(dVxdt); 
            __device_max_y(dVydt); 
            iterror = max(device_MAXx, device_MAXy);

            if(!(iterror>0 || iterror==0 || iterror<0)){printf("\n !! ERROR: err_MAX=Nan \n\n");break;} 
        
            std::cout<<"iter="<<iter<<", err="<<iterror<<std::endl;
            if ((iterror < epsi) && (iter > 100)) break;
        }
	
	}
    
     time_s = toc(); double gbs = mem/time_s;

    std::cout<<"Perf: "<<time_s<<" sec. (@ "<<gbs*(iter-10)*nIO<<" GB/s)"<<std::endl;
	
        /*Copy results from Device to host*/
	hipMemcpy(vx, d_vx, nbv*sizeof(double), hipMemcpyDeviceToHost );
	hipMemcpy(vy, d_vy, nbv*sizeof(double), hipMemcpyDeviceToHost ); 
        
	std::cout<<"iter="<<iter<<", err="<<iterror<<std::endl;

	/*Write output*/
	fid = fopen(outputfile,"wb");
	if(fid==NULL) std::cerr<<"could not open file " << outputfile << " for binary reading or writing";
	WriteData(fid,"PTsolution","SolutionType");
	WriteData(fid,vx,nbv,1,"Vx");
	WriteData(fid,vy,nbv,1,"Vy");
	if(fclose(fid)!=0) std::cerr<<"could not close file " << outputfile;
	

	/*Cleanup and return*/
	delete [] index;
	delete [] x;
	delete [] y;
	delete [] H;
	delete [] surface;
	delete [] base;
	delete [] spcvx;
	delete [] spcvy;
	delete [] ice_levelset;
	delete [] ocean_levelset;
	delete [] rheology_B;
	delete [] rheology_B_temp;
	delete [] vx;
	delete [] vy;
	delete [] friction;
	delete [] alpha2;
	delete [] etan;
	delete [] dVxdt;
	delete [] dVydt;
	delete [] alpha;
	delete [] beta;
	delete [] areas;
	delete [] weights;
	delete [] resolx;
	delete [] resoly;
	delete [] dsdx;
	delete [] dsdy;
	delete [] Helem;
	delete [] ML;
	delete [] Fvx;
	delete [] Fvy;

        hipFree(d_index);
	hipFree(d_vx);
	hipFree(d_vy);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_etan);
	hipFree(d_rheology_B);
	hipFree(d_Helem);
	hipFree(d_areas);
	hipFree(d_weights);
	hipFree(d_ML);
	hipFree(d_Fvx);
	hipFree(d_Fvy);
	hipFree(d_dVxdt);
	hipFree(d_dVydt);
	hipFree(d_resolx);
	hipFree(d_resoly);
	hipFree(d_H);
	hipFree(d_spcvx);
	hipFree(d_spcvy);   
	hipFree(d_alpha2);
	hipFree(d_groundedratio);
        hipFree(d_isice);
        hipFree(d_connectivity);
        hipFree(d_columns);        
        hipFree(d_device_normvalx);
        hipFree(d_device_normvaly);
        hipFree(d_device_maxvalx);
        hipFree(d_device_maxvaly);
	
	hipFree(dvxdx);
	hipFree(dvxdy);
	hipFree(dvydx);
	hipFree(dvydy);
	hipFree(KVx);
	hipFree(KVy);
	hipFree(eta_nbv);
	hipFree(Eta_nbe);
	hipFree(kvx);
        hipFree(kvy);
	
	//Destroying CUDA streams
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);


	return 0;
}/*}}}*/
