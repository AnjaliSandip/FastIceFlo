

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "sys/time.h"

#define GPU_ID 0

// #define USE_SINGLE_PRECISION    /* Comment this line using "!" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT     float
#define PRECIS  4
#else
#define DAT     double
#define PRECIS  8
#endif
#define zeros(A,nx,ny,nz)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc((nx)*(ny)*(nz)*sizeof(DAT)); \
                           for(i=0; i < (nx)*(ny)*(nz); i++){ A##_h[i]=(DAT)0.0; }              \
                           hipMalloc(&A##_d      ,(nx)*(ny)*(nz)*sizeof(DAT));                 \
                           hipMemcpy( A##_d,A##_h,(nx)*(ny)*(nz)*sizeof(DAT),hipMemcpyHostToDevice);
#define  free_all(A)       free(A##_h);hipFree(A##_d);

#define BLOCK_X   32
#define BLOCK_Y   16
#define BLOCK_Z   2
#define GRID_X    32
#define GRID_Y    64
#define GRID_Z    128
 
const int nx = GRID_X*BLOCK_X;
const int ny = GRID_Y*BLOCK_Y;
const int nz = GRID_Z*BLOCK_Z;
const int nt = 100;

// Timer
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }

void timPrint(const char *what, double n, int nx, int ny, int nz){
  double s=toc();
  printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n");
  FILE*fid; fid=fopen("PERF_memcpy.dat","a"); fprintf(fid,"nx=%d ny=%d nz=%d GBs=%1.4f  time_s=%1.4f \n", nx, ny, nz, n/s, s); fclose(fid);
}

void  clean_cuda(){ 
  hipError_t ce = hipGetLastError();
  if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset();}
}

__global__ void memcopy(DAT*A, DAT*B, DAT*C, const int nx,const int ny,const int nz){  
  int ix  = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
  int iy  = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension x
  int iz  = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension x
  if (iz<nz && iy<ny && ix<nx) A[ix + iy*nx + iz*nx*ny] = B[ix + iy*nx + iz*nx*ny] + C[ix + iy*nx + iz*nx*ny];
}

////////// main //////////
int main(){
  size_t i, it, N=nx*ny*nz, mem=3*N*sizeof(DAT);
  dim3 grid, block;
  block.x = BLOCK_X; block.y = BLOCK_Y; block.z = BLOCK_Z;
  grid.x  = GRID_X;  grid.y  = GRID_Y;  grid.z  = GRID_Z;
  int gpu_id=-1; gpu_id=GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
  hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
  printf("Process uses GPU with id %d.\n",gpu_id);
  printf("%dx%dx%d, %1.3f GB, %d iterations.\n", nx,ny,nz, mem/1024./1024./1024., nt);
  printf("launching (%dx%dx%d) grid of (%dx%dx%d) blocks.\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
  // initializations  
  zeros(A, nx,ny,nz);
  zeros(B, nx,ny,nz);
  zeros(C, nx,ny,nz);
  // time loop
  for(it=0; it<nt; it++){ 
    if (it==10){ tic(); }       
    memcopy<<<grid, block>>>(A_d,B_d,C_d,nx,ny,nz);
    hipDeviceSynchronize();
  }
  // tim("Performance", mem*(nt-3)*2/1024./1024./1024.);
  timPrint("Performance", mem*(nt-10)/1024./1024./1024.,nx,ny,nz);
  free_all(A);
  free_all(B);
  free_all(C);
  clean_cuda();
  return 0;
}
