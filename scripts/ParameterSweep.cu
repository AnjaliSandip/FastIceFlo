#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <string.h>
#include <cmath>
using namespace std;

/*define GPU specific variables*/
#define GPU_ID    6

#define BLOCK_Xe  256   //optimal block size for JKS2e4 and PIG3e4
#define BLOCK_Xv  256 

// Device norm subroutine
#define blockId       (blockIdx.x)
#define threadId      (threadIdx.x)
#define isBlockMaster (threadIdx.x==0)

#include "helpers.h"

/*CUDA Code*/
__global__ void PT1(double* vx, double* vy, double* alpha, double* beta, int* index,  double* kvx, double* kvy, double* etan,  double* Helem, double* areas, bool* isice, double* Eta_nbe, double* rheology_B, double n_glen, double eta_0, double rele,int nbe){ 
    // int ix = blockIdx.x * blockDim.x + threadIdx.x;
    for(int ix = blockIdx.x * blockDim.x + threadIdx.x; ix<nbe; ix += blockDim.x * gridDim.x){ 

		double Localalpha[3];
		double Localbeta[3];
		double Localvx[3];
		double Localvy[3];
    
        for(int i=0; i<3; i++){
            Localalpha[i] =  alpha[ix*3+i];
            Localbeta[i] =   beta[ix*3+i];
            Localvx[i] =  vx[index[ix*3+i]-1];
            Localvy[i] =  vy[index[ix*3+i]-1];
        }
       
        double dvxdx =  Localvx[0]*Localalpha[0] + Localvx[1]*Localalpha[1] + Localvx[2]*Localalpha[2];
        double dvxdy =  Localvx[0]* Localbeta[0] + Localvx[1]* Localbeta[1] + Localvx[2]* Localbeta[2];
        double dvydx =  Localvy[0]*Localalpha[0] + Localvy[1]*Localalpha[1] + Localvy[2]*Localalpha[2];
        double dvydy =  Localvy[0]* Localbeta[0] + Localvy[1]* Localbeta[1] + Localvy[2]* Localbeta[2];

        double  eps_xx = dvxdx;
        double  eps_yy = dvydy;
        double  eps_xy = .5*(dvxdy+dvydx);
        double  EII2   = eps_xx*eps_xx + eps_yy*eps_yy + eps_xy*eps_xy + eps_xx*eps_yy;
        double  eta_it = 1.e+14/2.0;

        if (EII2>0.) eta_it = rheology_B[ix]/(2*pow(EII2,(n_glen-1.)/(2*n_glen)));
            

       
        /*Skip if no ice*/
        if (isice[ix]){
            etan[ix] = min(exp(rele*log(eta_it) + (1-rele)*log(etan[ix])),eta_0*1e5);
            /*Viscous Deformation*/
            for (int i = 0; i < 3; i++){
                    kvx[ix*3+i] = 2 * Helem[ix] * etan[ix] *areas[ix]* (  (2 * eps_xx + eps_yy) * Localalpha[i]   +  eps_xy *  Localbeta[i] );
                    kvy[ix*3+i] = 2 * Helem[ix] *  etan[ix] * areas[ix] * ((2 * eps_yy + eps_xx) *  Localbeta[i]  +  eps_xy * Localalpha[i] );         
     }
        }//isice loop

   
        Eta_nbe[ix] = etan[ix]*areas[ix];
    }

}

//Moving to the next kernel, as kvx cannot be defined and updated in the same kernel
__global__ void PT2_x(double* kvx, double* groundedratio, double* areas, int* index, double* alpha2, double* vx, bool* isice,  int nbe){

    for(int ix = blockIdx.x * blockDim.x + threadIdx.x; ix < nbe; ix += blockDim.x * gridDim.x){
        /*Add basal friction*/
        if (groundedratio[ix] > 0.){
            int n3 = ix * 3;
            double gr_a = groundedratio[ix] * areas[ix];

       double myLocalIndex[3][3];
            for (int i = 0; i < 3; i++){
                int i_index = index[n3 + i] - 1;
                double gr_a_alpha2 =  gr_a * alpha2[i_index];
              	for (int j = 0; j < 3; j++){
                      int j_index = index[n3 + j] - 1;
                      myLocalIndex[i][j] = gr_a_alpha2 * vx[j_index];
                }
            }


            double tempOutput[3];
            for (int k = 0; k < 3; k++){
                tempOutput[k] = kvx[n3 + k];
            }   


            for (int k = 0; k < 3; k++){
                for (int i = 0; i < 3; i++){
                //    int i_index = index[n3 + i] - 1;
                //    double gr_a_alpha2 = gr_a * alpha2[i_index];
                    for (int j = 0; j < 3; j++){
                  //      int j_index = index[n3 + j] - 1;
                  //      double gr_a_alpha2_vx = gr_a_alpha2 * vx[j_index];
                        // printf("%d, %f, %f, %d, %f \n", ix, gr_a, gr_a_alpha2, j_index, gr_a_alpha2_vx);
                        	double temp = myLocalIndex[i][j];
                        if (i == j && j == k){
                            tempOutput[k]  = isice[ix] * tempOutput[k] + temp / 10.;
                        } else if ((i!=j) && (j!=k) && (k!=i)){
                            tempOutput[k] = isice[ix] * tempOutput[k] + temp / 60.;
                        } else{
                            tempOutput[k] = isice[ix] * tempOutput[k] + temp / 30.;
                        }
                    }
                }
            } 

             for (int k = 0; k < 3; k++){
                kvx[n3 + k] = tempOutput[k];
            }  
        }//groundedratio loop
    }
}

__global__ void PT2_y(double* kvy, double* groundedratio, double* areas, int* index, double* alpha2, double* vy, bool* isice,  int nbe){

   for(int ix = blockIdx.x * blockDim.x + threadIdx.x; ix < nbe; ix += blockDim.x * gridDim.x){
        /*Add basal friction*/
        if (groundedratio[ix] > 0.){
            int n3 = ix * 3;
            double gr_a = groundedratio[ix] * areas[ix];

            double myLocalIndex[3][3];
            for (int i = 0; i < 3; i++){
                int i_index = index[n3 + i] - 1;
                double gr_a_alpha2 =  gr_a * alpha2[i_index];
                for (int j = 0; j < 3; j++){
                      int j_index = index[n3 + j] - 1;
                      myLocalIndex[i][j] = gr_a_alpha2 * vy[j_index];
                }
            }

          double tempOutput[3];
            for (int k = 0; k < 3; k++){
                tempOutput[k] = kvy[n3 + k];
            }

            for (int k = 0; k < 3; k++){
                for (int i = 0; i < 3; i++){
                //   int i_index = index[n3 + i] - 1;
                //    double gr_a_alpha2 = gr_a * alpha2[i_index];
                    for (int j = 0; j < 3; j++){
                       // int j_index = index[n3 + j] - 1;
                      //  double gr_a_alpha2_vy = gr_a_alpha2 * vy[j_index];
                        // printf("%d, %f, %f, %d, %f \n", ix, gr_a, gr_a_alpha2, j_index, gr_a_alpha2_vx);
                     double temp = myLocalIndex[i][j];
                        if (i == j && j == k){
                            tempOutput[k] = isice[ix] * tempOutput[k] + temp / 10.;
                        } else if ((i!=j) && (j!=k) && (k!=i)){
                            tempOutput[k] = isice[ix] * tempOutput[k] + temp / 60.;
                        } else{
                            tempOutput[k] = isice[ix] * tempOutput[k] + temp / 30.;
                        }
                    }
                }
            } 

              for (int k = 0; k < 3; k++){
                kvy[n3 + k] = tempOutput[k];
            }  
  

        }//groundedratio loop
    }
}

//Moving to the next kernel: cannot update kvx and perform indirect access, lines 474 and 475, in the same kernel//
__global__ void PT3(double* kvx, double* kvy, double* Eta_nbe, double* areas, double* eta_nbv, int* index, int* connectivity, int* columns, double* weights, double* ML, double* KVx, double* KVy, double* Fvx, double* Fvy, double* dVxdt, double* dVydt, double* resolx, double* resoly, double* H, double* vx, double* vy, double* spcvx, double* spcvy, double rho, double damp, double relaxation, double eta_b, int nbv){ 

    double ResVx;
    double ResVy;
    double dtVx;
    double dtVy;

    for(int ix = blockIdx.x * blockDim.x + threadIdx.x; ix<nbv; ix += blockDim.x * gridDim.x){

        KVx[ix] = 0.;
        KVy[ix] = 0.;

/*        for(int j=0;j<8;j++){
       //     if (connectivity[(ix * 8 + j)] != 0){
                   if (connectivity[((ix << 3) + j)] != 0){
                KVx[ix] = KVx[ix] + kvx[((connectivity[(ix * 8 + j)])-1) *3 + ((columns[(ix * 8 + j)]))];
                KVy[ix] = KVy[ix] + kvy[((connectivity[(ix * 8 + j)])-1) *3 + ((columns[(ix * 8 + j)]))];
            }
        }

        for (int j = 0; j < 8; j++){
            if (connectivity[((ix << 3) + j)] != 0){
                eta_nbv[ix] = eta_nbv[ix] + Eta_nbe[connectivity[(ix * 8 + j)]-1];
            }
        }  
    
*/


				int localConnectivity[8];
        for(int j=0; j<8; j++){
	        localConnectivity[j] = connectivity[(ix * 8  + j)];
        }


        for(int j=0;j<8;j++){
        
            if (localConnectivity[j] != 0){
                KVx[ix] = KVx[ix] + kvx[((localConnectivity[j])-1) *3 + ((columns[(ix * 8  + j)]))];
                KVy[ix] = KVy[ix] + kvy[((localConnectivity[j])-1) *3 + ((columns[(ix * 8 +  j)]))];
            }
        }

        for (int j = 0; j < 8; j++){
            if (localConnectivity[j] != 0){
                eta_nbv[ix] = eta_nbv[ix] + Eta_nbe[localConnectivity[j]-1];
            }
        }    


        eta_nbv[ix] =eta_nbv[ix]/weights[ix];

        /*1. Get time derivative based on residual (dV/dt)*/
        ResVx =  (-KVx[ix] + Fvx[ix])/(rho*max(60.0,H[ix])*ML[ix]);      
        ResVy =  (-KVy[ix] + Fvy[ix])/(rho*max(60.0,H[ix])*ML[ix]);

        dVxdt[ix] = dVxdt[ix]*damp + ResVx;
        dVydt[ix] = dVydt[ix]*damp + ResVy;

    
        /*2. Explicit CFL time step for viscous flow, x and y directions*/
        dtVx = rho*resolx[ix]*resolx[ix]/(4*eta_nbv[ix]*(1.+eta_b)*4.1);
        dtVy = rho*resoly[ix]*resoly[ix]/(4*eta_nbv[ix]*(1.+eta_b)*4.1);

        /*3. velocity update, vx(new) = vx(old) + change in vx, Similarly for vy*/
        vx[ix] = vx[ix] + relaxation*dVxdt[ix]*dtVx;
        vy[ix] = vy[ix] + relaxation*dVydt[ix]*dtVy;

        /*Apply Dirichlet boundary condition*/
        if (!isnan(spcvx[ix])){
            vx[ix]    = spcvx[ix];
            dVxdt[ix] = 0.;
        }
        if (!isnan(spcvy[ix])){
            vy[ix]    = spcvy[ix];
            dVydt[ix] = 0.;
        }
    }
}

/*Main*/
int main(){
           /*If param sweep*/
    bool do_save = DO_SAVE;
    const char* outfile = "./output.txt";


      /*Open input binary file*/
    const char* inputfile  = "./JKS8e4.bin";
    const char* outputfile = "./output.outbin";
    FILE* fid = fopen(inputfile,"rb");
    if(fid==NULL) std::cerr<<"could not open file " << inputfile << " for binary reading or writing";


    /*Get All we need from binary file*/
    int    nbe,nbv,M,N;
    double g,rho,rho_w,yts;
    int    *index           = NULL;
    double *spcvx           = NULL;
    double *spcvy           = NULL;
    double *x               = NULL;
    double *y               = NULL;
    double *H               = NULL;
    double *surface         = NULL;
    double *base            = NULL;
    double *ice_levelset    = NULL;
    double *ocean_levelset  = NULL;
    double *rheology_B_temp = NULL;
    double *vx              = NULL;
    double *vy              = NULL;
    double *friction        = NULL;
    FetchData(fid,&nbe,"md.mesh.numberofelements");
    FetchData(fid,&nbv,"md.mesh.numberofvertices");
    FetchData(fid,&g,"md.constants.g");
    FetchData(fid,&rho,"md.materials.rho_ice");
    FetchData(fid,&rho_w,"md.materials.rho_water");
    FetchData(fid,&yts,"md.constants.yts");
    FetchData(fid,&index,&M,&N,"md.mesh.elements");
    FetchData(fid,&spcvx,&M,&N,"md.stressbalance.spcvx");
    FetchData(fid,&spcvy,&M,&N,"md.stressbalance.spcvy");
    FetchData(fid,&x,&M,&N,"md.mesh.x");
    FetchData(fid,&y,&M,&N,"md.mesh.y");
    FetchData(fid,&H,&M,&N,"md.geometry.thickness");
    FetchData(fid,&surface,&M,&N,"md.geometry.surface");
    FetchData(fid,&base,&M,&N,"md.geometry.base");
    FetchData(fid,&ice_levelset,&M,&N,"md.mask.ice_levelset");
    FetchData(fid,&ocean_levelset,&M,&N,"md.mask.ocean_levelset");
    FetchData(fid,&rheology_B_temp,&M,&N,"md.materials.rheology_B");
    FetchData(fid,&vx,&M,&N,"md.initialization.vx");
    FetchData(fid,&vy,&M,&N,"md.initialization.vy");
    FetchData(fid,&friction,&M,&N,"md.friction.coefficient");

    /*Close input file*/
    if(fclose(fid)!=0) std::cerr<<"could not close file " << inputfile;

    /*Constants*/
    double n_glen     = 3.;
    double damp       = dmp; //0.96 for JKS2e4, 0.981 for PIG3e4
    double rele       = 1e-1;   //1e-1 for JKS2e4, 0.07 for PIG3e4
    double eta_b      = 0.5;
    double eta_0      = 1.e+14/2.;
    int    niter      = 5e6; //5e6
    int    nout_iter  = 100; //change it to 100 for JKS2e4
    double epsi       = 3.171e-7;
    double relaxation = rela; //0.7 for JKS2e4, 0.967 for PIG3e4
        
    // Ceiling division to get the close to optimal GRID size
    unsigned int GRID_Xe = 1 + ((nbe - 1) / BLOCK_Xe);
    unsigned int GRID_Xv = 1 + ((nbv - 1) / BLOCK_Xv);

    GRID_Xe = GRID_Xe - GRID_Xe%80;
    GRID_Xv = GRID_Xv - GRID_Xv%80;

    std::cout<<"GRID_Xe="<<GRID_Xe<<std::endl;
    std::cout<<"GRID_Xv="<<GRID_Xv<<std::endl;

    // Set up GPU
    int gpu_id=-1;
    dim3 gridv, blockv;
    dim3 gride, blocke;
    blockv.x = BLOCK_Xv; gridv.x = GRID_Xv;
    blocke.x = BLOCK_Xe; gride.x = GRID_Xe;
    gpu_id = GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n", gpu_id);
    //hipSetDevice  selects the device, set the gpu id you selected

    /*Initial guesses (except vx and vy that we already loaded)*/
    double* etan = new double[nbe];
    for(int i=0;i<nbe;i++) etan[i] = 1.e+14;
    double* dVxdt = new double[nbv];
    for(int i=0;i<nbv;i++) dVxdt[i] = 0.;
    double* dVydt = new double[nbv];
    for(int i=0;i<nbv;i++) dVydt[i] = 0.;

    /*Manage derivatives once for all*/
    double* alpha   = NULL;
    double* beta    = NULL;
    double* areas   = NULL;
    double* weights = NULL;
    NodalCoeffs(&areas,&alpha,&beta,index,x,y,nbe);
    Weights(&weights,index,areas,nbe,nbv);

    /*MeshSize*/
    double* resolx = new double[nbv];
    double* resoly = new double[nbv];
    MeshSize(resolx,resoly,index,x,y,areas,weights,nbe,nbv);

    /*Physical properties once for all*/
    double* dsdx = new double[nbe];
    double* dsdy = new double[nbe];
    derive_xy_elem(dsdx,dsdy,surface,index,alpha,beta,nbe);
    double* Helem      = new double[nbe];
    double* rheology_B = new double[nbe];
    for(int i=0;i<nbe;i++){
        Helem[i]      = 1./3. * (H[index[i*3+0]-1] + H[index[i*3+1]-1] + H[index[i*3+2]-1]);
        rheology_B[i] = 1./3. * (rheology_B_temp[index[i*3+0]-1] + rheology_B_temp[index[i*3+1]-1] + rheology_B_temp[index[i*3+2]-1]);
    }

    //Initial viscosity//
    double* dvxdx   = new double[nbe];
    double* dvxdy   = new double[nbe];
    double* dvydx   = new double[nbe];
    double* dvydy   = new double[nbe];

    derive_xy_elem(dvxdx,dvxdy,vx,index,alpha,beta,nbe);
    derive_xy_elem(dvydx,dvydy,vy,index,alpha,beta,nbe);

    for(int i=0;i<nbe;i++){
        double eps_xx = dvxdx[i];
        double eps_yy = dvydy[i];
        double eps_xy = .5*(dvxdy[i]+dvydx[i]);
        double EII2 = pow(eps_xx,2) + pow(eps_yy,2) + pow(eps_xy,2) + eps_xx*eps_yy;
        double eta_it = 1.e+14/2.;
        if (EII2>0.) eta_it = rheology_B[i]/(2*pow(EII2,(n_glen-1.)/(2*n_glen)));

        etan[i] = min(eta_it,eta_0*1e5);
        if (isnan(etan[i])){ std::cerr<<"Found NaN in etan[i]"; return 1;}
    }

    /*Linear integration points order 3*/
    double wgt3[] = { 0.555555555555556, 0.888888888888889, 0.555555555555556 };
    double xg3[]  = {-0.774596669241483, 0.000000000000000, 0.774596669241483 };

    /*Compute RHS amd ML once for all*/
    double* ML            = new double[nbv];
    double* Fvx           = new double[nbv];
    double* Fvy           = new double[nbv];
    double* groundedratio = new double[nbe];
    bool*   isice         = new bool[nbe];     
    double level[3];      

    for(int i=0;i<nbv;i++){
        ML[i]  = 0.;
        Fvx[i] = 0.;
        Fvy[i] = 0.;
    }
    for(int n=0;n<nbe;n++){
        /*Lumped mass matrix*/
        for(int i=0;i<3;i++){
            for(int j=0;j<3;j++){
                // \int_E phi_i * phi_i dE = A/6 and % \int_E phi_i * phi_j dE = A/12
                if (i==j)
                 ML[index[n*3+j]-1] += areas[n]/6.;
                else
                 ML[index[n*3+j]-1] += areas[n]/12.;
            }
        }
        /*Is there ice at all in the current element?*/
        level[0] = ice_levelset[index[n*3+0]-1];
        level[1] = ice_levelset[index[n*3+1]-1];
        level[2] = ice_levelset[index[n*3+2]-1];
        if (level[0]<0 || level[1]<0 || level[2]<0){
            isice[n] = true;
        }
        else{
            isice[n] = false;
            for(int i=0;i<3;i++){
                vx[index[n*3+i]-1] = 0.;
                vy[index[n*3+i]-1] = 0.;
            }
            continue;
        }
        /*RHS, 'F ' in equation 22 (Driving Stress)*/
        for(int i=0;i<3;i++){
            for(int j=0;j<3;j++){
                if (i==j){
                    Fvx[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdx[n]*areas[n]/6.;
                    Fvy[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdy[n]*areas[n]/6.;
                }
                else{
                    Fvx[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdx[n]*areas[n]/12.;
                    Fvy[index[n*3+i]-1] += -rho*g*H[index[n*3+j]-1]*dsdy[n]*areas[n]/12.;
                }
            }
        }
    }

    /*RHS (Water pressure at the ice front)*/
    //  double level[3];
    for(int n=0;n<nbe;n++){
        /*Determine if there is an ice front there*/
        level[0] = ice_levelset[index[n*3+0]-1];
        level[1] = ice_levelset[index[n*3+1]-1];
        level[2] = ice_levelset[index[n*3+2]-1];
        int count = 0;
        for(int i=0;i<3;i++) if (level[i]<0.) count++;
        if (count==1){
            /*Ok this element has an ice front, get indices of the 2 vertices*/
            int seg1[2] = {index[n*3+0]-1,index[n*3+1]-1};
            int seg2[2] = {index[n*3+1]-1,index[n*3+2]-1};
            int seg3[2] = {index[n*3+2]-1,index[n*3+0]-1};
            int pairids[2];
            if (ice_levelset[seg1[0]]>=0 && ice_levelset[seg1[1]]>=0){
                pairids[0] = seg1[0]; pairids[1] = seg1[1];
            }
            else if (ice_levelset[seg2[0]]>=0 && ice_levelset[seg2[1]]>=0){
                pairids[0] = seg2[0]; pairids[1] = seg2[1];
            }
            else if (ice_levelset[seg3[0]]>=0 && ice_levelset[seg3[1]]>=0){
                pairids[0] = seg3[0]; pairids[1] = seg3[1];
            }
            else{
                std::cerr<<"case not supported";
            }
            /*Get normal*/
            double len = sqrt(pow(x[pairids[1]]-x[pairids[0]],2) + pow(y[pairids[1]]-y[pairids[0]],2) );
            double nx  = +(y[pairids[1]]-y[pairids[0]])/len;
            double ny  = -(x[pairids[1]]-x[pairids[0]])/len;
            /*RHS*/
            for(int gg=0;gg<2;gg++){
                double phi1 = (1.0 -xg3[gg])/2.;
                double phi2 = (1.0 +xg3[gg])/2.;
                double bg = base[pairids[0]]*phi1 + base[pairids[1]]*phi2;
                double Hg = H[pairids[0]]*phi1 + H[pairids[1]]*phi2;
                bg = min(bg,0.0);
                Fvx[pairids[0]] = Fvx[pairids[0]] +wgt3[gg]/2*1/2*(-rho_w*g* pow(bg,2)+rho*g*pow(Hg,2))*nx*len*phi1;
                Fvx[pairids[1]] = Fvx[pairids[1]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*nx*len*phi2;
                Fvy[pairids[0]] = Fvy[pairids[0]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*ny*len*phi1;
                Fvy[pairids[1]] = Fvy[pairids[1]] +wgt3[gg]/2*1/2*(-rho_w*g*pow(bg,2)+rho*g*pow(Hg,2))*ny*len*phi2;
            } 
        }
        /*One more thing in this element loop: prepare groundedarea needed later for the calculation of basal friction*/
        level[0] = ocean_levelset[index[n*3+0]-1];
        level[1] = ocean_levelset[index[n*3+1]-1];
        level[2] = ocean_levelset[index[n*3+2]-1];
        if (level[0]>=0. && level[1]>=0. && level[2]>=0.){
            /*Completely grounded*/
            groundedratio[n]=1.;
        }
        else if (level[0]<=0. && level[1]<=0. && level[2]<=0.){
            /*Completely floating*/
            groundedratio[n]=0.;
        }
        else{
            /*Partially floating,*/
            double s1,s2;
            if (level[0]*level[1]>0){/*Nodes 0 and 1 are similar, so points must be found on segment 0-2 and 1-2*/
                s1=level[2]/(level[2]-level[1]);
                s2=level[2]/(level[2]-level[0]);
            }
            else if (level[1]*level[2]>0){ /*Nodes 1 and 2 are similar, so points must be found on segment 0-1 and 0-2*/
                s1=level[0]/(level[0]-level[1]);
                s2=level[0]/(level[0]-level[2]);
            }
            else if (level[0]*level[2]>0){/*Nodes 0 and 2 are similar, so points must be found on segment 1-0 and 1-2*/
                s1=level[1]/(level[1]-level[0]);
                s2=level[1]/(level[1]-level[2]);
            }
            else{
                std::cerr<<"should not be here...";
            }

            if (level[0]*level[1]*level[2]>0.){
                /*two nodes floating, inner triangle is grounded*/
                groundedratio[n]= s1*s2;
            }
            else{
                /*one node floating, inner triangle is floating*/
                groundedratio[n]= (1.-s1*s2);
            }
        }
    }

    /*Finally add calculation of friction coefficient*/
    double* alpha2 = new double[nbv];
    for(int i=0;i<nbv;i++){
        /*Compute effective pressure*/
        double p_ice   = g*rho*H[i];
        double p_water = -rho_w*g*base[i];
        double Neff    = p_ice - p_water;
        if (Neff<0.) Neff=0.;
        /*Compute alpha2*/
        alpha2[i] = pow(friction[i],2)*Neff;
    }

    //prepare head and next vectors for chain algorithm, at this point we have not seen any of the elements, so just set the head to -1 (=stop)
    int* head = new int[nbv];
    int* next  = new int[3*nbe];
    for(int i=0;i<nbv;i++) head[i] = -1;

    //Now construct the chain
    for(int k=0;k<nbe;k++){
        for(int j=0;j<3;j++){
            int i;
            int p = 3*k+j;       //unique linear index of current vertex in index
            i = index[p];
            next[p] = head[i - 1];
            head[i -1] = p + 1;
        }
    }

    //Note: Index array starts at 0, but the node# starts at 1
    //Now we can construct the connectivity matrix
    int MAXCONNECT = 8;
    int* connectivity = new int[nbv*MAXCONNECT];
    int* columns = new int[nbv*MAXCONNECT];

    for(int i=0;i<nbv;i++){

        /*Go over all of the elements connected to node I*/
        int count = 0;
        int p=head[i];

        //for (int p = head[i]; p != -1; p = next[p]){
        while (p!= -1){

            int k = p / 3 + 1;     //”row" in index
            int j = (p % 3) - 1;   //"column" in index

            if (j==-1){
                j=2;
                k= k -1;}

            //sanity check
            if (index[p-1] !=i+1){
                std::cout << "Error occurred"  << std::endl;;
            }

            //enter element in connectivity matrix
            connectivity[i * MAXCONNECT + count] = k;
            columns[i * MAXCONNECT + count] = j;
            count++;
            p = next[p-1];
        }
    }

    double* device_maxvalx = new double[GRID_Xv];
    double* device_maxvaly = new double[GRID_Xv];
    for(int i=0;i<GRID_Xv;i++) device_maxvalx[i] = 0.;
    for(int i=0;i<GRID_Xv;i++) device_maxvaly[i] = 0.;

    /*------------ now copy all relevant vectors from host to device ---------------*/
    int *d_index = NULL;
    hipMalloc(&d_index, nbe*3*sizeof(int));
    hipMemcpy(d_index, index, nbe*3*sizeof(int), hipMemcpyHostToDevice);

    double *d_vx;
    hipMalloc(&d_vx, nbv*sizeof(double));
    hipMemcpy(d_vx, vx, nbv*sizeof(double), hipMemcpyHostToDevice);  

    double *d_vy;
    hipMalloc(&d_vy, nbv*sizeof(double));
    hipMemcpy(d_vy, vy, nbv*sizeof(double), hipMemcpyHostToDevice);  

    double *d_alpha;
    hipMalloc(&d_alpha, nbe*3*sizeof(double));
    hipMemcpy(d_alpha, alpha, nbe*3*sizeof(double), hipMemcpyHostToDevice);

    double *d_beta;
    hipMalloc(&d_beta, nbe*3*sizeof(double));
    hipMemcpy(d_beta, beta, nbe*3*sizeof(double), hipMemcpyHostToDevice);

    double *d_etan;
    hipMalloc(&d_etan, nbe*sizeof(double));
    hipMemcpy(d_etan, etan, nbe*sizeof(double), hipMemcpyHostToDevice);  

    double *d_rheology_B;
    hipMalloc(&d_rheology_B, nbe*sizeof(double));
    hipMemcpy(d_rheology_B, rheology_B, nbe*sizeof(double), hipMemcpyHostToDevice); 

    double *d_Helem;
    hipMalloc(&d_Helem, nbe*sizeof(double));
    hipMemcpy(d_Helem, Helem, nbe*sizeof(double), hipMemcpyHostToDevice); 

    double *d_areas;
    hipMalloc(&d_areas, nbe*sizeof(double));
    hipMemcpy(d_areas, areas, nbe*sizeof(double), hipMemcpyHostToDevice); 

    double *d_weights;
    hipMalloc(&d_weights, nbv*sizeof(double));
    hipMemcpy(d_weights, weights, nbv*sizeof(double), hipMemcpyHostToDevice);  

    double *d_ML;
    hipMalloc(&d_ML, nbv*sizeof(double));
    hipMemcpy(d_ML, ML, nbv*sizeof(double), hipMemcpyHostToDevice);  

    double *d_Fvx;
    hipMalloc(&d_Fvx, nbv*sizeof(double));
    hipMemcpy(d_Fvx, Fvx, nbv*sizeof(double), hipMemcpyHostToDevice); 

    double *d_Fvy;
    hipMalloc(&d_Fvy, nbv*sizeof(double));
    hipMemcpy(d_Fvy, Fvy, nbv*sizeof(double), hipMemcpyHostToDevice); 

    double *d_dVxdt;
    hipMalloc(&d_dVxdt, nbv*sizeof(double));
    hipMemcpy(d_dVxdt, dVxdt, nbv*sizeof(double), hipMemcpyHostToDevice); 

    double *d_dVydt;
    hipMalloc(&d_dVydt, nbv*sizeof(double));
    hipMemcpy(d_dVydt, dVydt, nbv*sizeof(double), hipMemcpyHostToDevice); 

    double *d_resolx;
    hipMalloc(&d_resolx, nbv*sizeof(double));
    hipMemcpy(d_resolx, resolx, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_resoly;
    hipMalloc(&d_resoly, nbv*sizeof(double));
    hipMemcpy(d_resoly, resoly, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_H;
    hipMalloc(&d_H, nbv*sizeof(double));
    hipMemcpy(d_H, H, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_spcvx;
    hipMalloc(&d_spcvx, nbv*sizeof(double));
    hipMemcpy(d_spcvx, spcvx, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_spcvy;
    hipMalloc(&d_spcvy, nbv*sizeof(double));
    hipMemcpy(d_spcvy, spcvy, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_alpha2;
    hipMalloc(&d_alpha2, nbv*sizeof(double));
    hipMemcpy(d_alpha2, alpha2, nbv*sizeof(double), hipMemcpyHostToDevice);

    double *d_groundedratio;
    hipMalloc(&d_groundedratio, nbe*sizeof(double));
    hipMemcpy(d_groundedratio, groundedratio, nbe*sizeof(double), hipMemcpyHostToDevice);

    bool *d_isice;
    hipMalloc(&d_isice, nbe*sizeof(bool));
    hipMemcpy(d_isice, isice, nbe*sizeof(bool), hipMemcpyHostToDevice);

    int *d_connectivity = NULL;
    hipMalloc(&d_connectivity, nbv*8*sizeof(int));
    hipMemcpy(d_connectivity, connectivity, nbv*8*sizeof(int), hipMemcpyHostToDevice);

    int *d_columns = NULL;
    hipMalloc(&d_columns, nbv*8*sizeof(int));
    hipMemcpy(d_columns, columns, nbv*8*sizeof(int), hipMemcpyHostToDevice);

    double* d_device_maxvalx = NULL;
    hipMalloc(&d_device_maxvalx, GRID_Xv*sizeof(double));
    hipMemcpy(d_device_maxvalx, device_maxvalx, GRID_Xv*sizeof(double), hipMemcpyHostToDevice);

    double* d_device_maxvaly = NULL;
    hipMalloc(&d_device_maxvaly, GRID_Xv*sizeof(double));
    hipMemcpy(d_device_maxvaly, device_maxvaly, GRID_Xv*sizeof(double), hipMemcpyHostToDevice); 

    /*------------ allocate relevant vectors on host (GPU)---------------*/
    //double *dvxdx = NULL;
    hipMalloc(&dvxdx,nbe*sizeof(double));

    //double *dvxdy = NULL;
    hipMalloc(&dvxdy, nbe*sizeof(double));

    //double *dvydx = NULL;
    hipMalloc(&dvydx, nbe*sizeof(double));

    //double *dvydy = NULL;
    hipMalloc(&dvydy, nbe*sizeof(double));

    double *KVx = NULL;
    hipMalloc(&KVx, nbv*sizeof(double));

    double *KVy = NULL;
    hipMalloc(&KVy, nbv*sizeof(double));

    double *eta_nbv = NULL;
    hipMalloc(&eta_nbv, nbv*sizeof(double));

    double *Eta_nbe = NULL;
    hipMalloc(&Eta_nbe, nbe*3*sizeof(double));

    double *kvx = NULL;
    hipMalloc(&kvx, nbe*3*sizeof(double));

    double *kvy = NULL;
    hipMalloc(&kvy, nbe*3*sizeof(double));
    
    //Creating CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    // Perf
    double time_s = 0.0;
    double mem = (double)1e-9*(double)nbv*sizeof(double);
    int nIO = 8;

    /*Main loop*/
    std::cout<<"Starting PT loop, nbe="<<nbe<<", nbv="<<nbv<<std::endl; 
    int iter;
    double iterror;
    for(iter=1;iter<=niter;iter++){
        
        if (iter==11) tic();

        PT1<<<gride, blocke>>>(d_vx, d_vy, d_alpha, d_beta, d_index, kvx,  kvy, d_etan, d_Helem, d_areas, d_isice, Eta_nbe, d_rheology_B, n_glen, eta_0, rele, nbe);
        hipDeviceSynchronize();

        PT2_x<<<gride, blocke, 0, stream1>>>(kvx, d_groundedratio, d_areas, d_index, d_alpha2, d_vx, d_isice, nbe);
//        hipStreamSynchronize(stream1);
        PT2_y<<<gride, blocke, 0, stream2>>>(kvy, d_groundedratio, d_areas, d_index, d_alpha2, d_vy, d_isice, nbe);
//        hipStreamSynchronize(stream2);
       

        PT3<<<gridv, blockv>>>(kvx, kvy, Eta_nbe, d_areas, eta_nbv, d_index, d_connectivity, d_columns, d_weights, d_ML, KVx, KVy, d_Fvx, d_Fvy, d_dVxdt, d_dVydt, d_resolx, d_resoly, d_H, d_vx, d_vy, d_spcvx, d_spcvy, rho, damp, relaxation, eta_b, nbv);
        hipDeviceSynchronize();

        if ((iter % nout_iter) == 0){
            //Get final error estimate/
            __device_max_x(dVxdt); 
            __device_max_y(dVydt); 
            iterror = max(device_MAXx, device_MAXy);
            if(!(iterror>0 || iterror==0 || iterror<0)){printf("\n !! ERROR: err_MAX=Nan \n\n");break;} 
            std::cout<<"iter="<<iter<<", err="<<iterror<<std::endl;
            if ((iterror < epsi) && (iter > 100)) break;
        }  
    }

    time_s = toc(); double gbs = mem/time_s;

    std::cout<<"Perf: "<<time_s<<" sec. (@ "<<gbs*(iter-10)*nIO<<" GB/s)"<<std::endl;

    /*Copy results from Device to host*/
    hipMemcpy(vx, d_vx, nbv*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(vy, d_vy, nbv*sizeof(double), hipMemcpyDeviceToHost );

    /*Write output*/
    fid = fopen(outputfile,"wb");
    if (fid==NULL) std::cerr<<"could not open file " << outputfile << " for binary reading or writing";
    WriteData(fid, "PTsolution", "SolutionType");
    WriteData(fid, vx, nbv, 1, "Vx");
    WriteData(fid, vy, nbv, 1, "Vy");
  //  WriteData(fid, sqrt(vx*vx+vy*vy)*yts, nbv, 1, "Vel");
    if (fclose(fid)!=0) std::cerr<<"could not close file " << outputfile;

    /*output to txt file*/
    if (do_save == true){
        fid = fopen(outfile, "a");
        fprintf(fid, "nbv=%d, nbe=%d, damp=%1.3f, relaxation =%1.3f,iters=%d \n", nbv, nbe, damp, rele, iter);
        fclose(fid);
    }


    /*Cleanup and return*/
    delete [] index;
    delete [] x;
    delete [] y;
    delete [] H;
    delete [] surface;
    delete [] base;
    delete [] spcvx;
    delete [] spcvy;
    delete [] ice_levelset;
    delete [] ocean_levelset;
    delete [] rheology_B;
    delete [] rheology_B_temp;
    delete [] vx;
    delete [] vy;
    delete [] friction;
    delete [] alpha2;
    delete [] etan;
    delete [] dVxdt;
    delete [] dVydt;
    delete [] alpha;
    delete [] beta;
    delete [] areas;
    delete [] weights;
    delete [] resolx;
    delete [] resoly;
    delete [] dsdx;
    delete [] dsdy;
    delete [] Helem;
    delete [] ML;
    delete [] Fvx;
    delete [] Fvy;

    hipFree(d_index);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_etan);
    hipFree(d_rheology_B);
    hipFree(d_Helem);
    hipFree(d_areas);
    hipFree(d_weights);
    hipFree(d_ML);
    hipFree(d_Fvx);
    hipFree(d_Fvy);
    hipFree(d_dVxdt);
    hipFree(d_dVydt);
    hipFree(d_resolx);
    hipFree(d_resoly);
    hipFree(d_H);
    hipFree(d_spcvx);
    hipFree(d_spcvy);   
    hipFree(d_alpha2);
    hipFree(d_groundedratio);
    hipFree(d_isice);
    hipFree(d_connectivity);
    hipFree(d_columns);
    hipFree(dvxdx);
    hipFree(dvxdy);
    hipFree(dvydx);
    hipFree(dvydy);
    hipFree(KVx);
    hipFree(KVy);
    hipFree(eta_nbv);
    hipFree(Eta_nbe);
    hipFree(kvx);
    hipFree(kvy);
    hipFree(d_device_maxvalx);
    hipFree(d_device_maxvaly);

    //Destroying CUDA streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    clean_cuda();
    return 0;
}
